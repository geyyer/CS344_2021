#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

// A common way to represent color images is known as RGBA - the color
// is specified by how much Red, Grean and Blue is in it.
// The 'A' stands for Alpha and is used for transparency, it will be
// ignored in this homework.

// Each channel Red, Blue, Green and Alpha is represented by one byte.
// Since we are using one byte for each color there are 256 different
// possible values for each color.  This means we use 4 bytes per pixel.

// Greyscale images are represented by a single intensity value per pixel
// which is one byte in size.

// To convert an image from color to grayscale one simple method is to
// set the intensity to the average of the RGB channels.  But we will
// use a more sophisticated method that takes into account how the eye
// perceives color and weights the channels unequally.

// The eye responds most strongly to green followed by red and then blue.
// The NTSC (National Television System Committee) recommends the following
// formula for color to greyscale conversion:

// I = .299f * R + .587f * G + .114f * B

// Notice the trailing f's on the numbers which indicate that they are
// single precision floating point constants and not double precision
// constants.

// You should fill in the kernel as well as set the block and grid sizes
// so that the entire image is processed.

#include "utils.h"

struct __align__(32) uchar32 {
  uchar4 x;
  uchar4 y;
  uchar4 z;
  uchar4 w;
  uchar4 a;
  uchar4 b;
  uchar4 c;
  uchar4 d;
};

struct __align__(8) uchar8 {
  unsigned char x;
  unsigned char y;
  unsigned char z;
  unsigned char w;
  unsigned char a;
  unsigned char b;
  unsigned char c;
  unsigned char d;
};

__global__ void rgba_to_greyscale(const uchar4 *const rgbaImage,
                                  unsigned char *const greyImage, 
                                  int numRows,
                                  int numCols) {

  int pixels = numRows * numCols;
  int eighth = pixels / 8;
  int limit = eighth * 8;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < eighth) {
    uchar32 load = ((const uchar32*) rgbaImage)[idx];
    uchar4 l0 = load.x;
    float tile0 = .299f * l0.x + .587f * l0.y + .114f * l0.z;
    uchar4 l1 = load.y;
    float tile1 = .299f * l1.x + .587f * l1.y + .114f * l1.z;
    uchar4 l2 = load.z;
    float tile2 = .299f * l2.x + .587f * l2.y + .114f * l2.z;
    uchar4 l3 = load.w;
    float tile3 = .299f * l3.x + .587f * l3.y + .114f * l3.z;
    uchar4 l4 = load.a;
    float tile4 = .299f * l4.x + .587f * l4.y + .114f * l4.z;
    uchar4 l5 = load.b;
    float tile5 = .299f * l5.x + .587f * l5.y + .114f * l5.z;
    uchar4 l6 = load.c;
    float tile6 = .299f * l6.x + .587f * l6.y + .114f * l6.z;
    uchar4 l7 = load.d;
    float tile7 = .299f * l7.x + .587f * l7.y + .114f * l7.z;

    uchar8 out;
    out.x = (unsigned char) tile0;
    out.y = (unsigned char) tile1;
    out.z = (unsigned char) tile2;
    out.w = (unsigned char) tile3;
    out.a = (unsigned char) tile4;
    out.b = (unsigned char) tile5;
    out.c = (unsigned char) tile6;
    out.d = (unsigned char) tile7;

    ((uchar8*) greyImage)[idx] = out;    
  }
  
  if (limit + idx < pixels) {
    uchar4 load = rgbaImage[limit + idx];
    float tile = .299f * load.x + .587f * load.y + .114f * load.z;
    greyImage[limit + idx] = (unsigned char) tile;
  }
}

void your_rgba_to_greyscale(const uchar4 *const h_rgbaImage,
                            uchar4 *const d_rgbaImage,
                            unsigned char *const d_greyImage, 
                            size_t numRows,
                            size_t numCols) {
  // You must fill in the correct sizes for the blockSize and gridSize
  // currently only one block with one thread is being launched
  const size_t pixels = numRows * numCols;
  const size_t thread_num = 64;
  const dim3 blockSize(thread_num, 1, 1);
  const dim3 gridSize((pixels + thread_num * 8 - 1) / (thread_num * 8), 1, 1);
  printf("Starting execution \n");
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, 
                                             d_greyImage, 
                                             numRows,
                                             numCols);

  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}
